#include "hip/hip_runtime.h"
#include "kan.h"

__global__ void kanKernel(float *y){

	// initialise random state
	hiprandStatePhilox4_32_10_t localRandState;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(seed, idx, 0, &localRandState);

    y[idx] = hiprand_uniform(&localRandState);
	// input x is of size batch_size,input_dim
	// output y is of size batch_size, output_dim
	// multiply with harmonics, randomly generated in a certain range

}

int main(){
	int blocks = 10000;
	int threads = 1024;
	int totalThreads = blocks*threads;
	float *hostResults = (float *)calloc(totalThreads, sizeof(float));
	float *devResults = (float *)calloc(totalThreads, sizeof(float));
    /* Allocate space for results on device */
    hipMalloc((void **)&devResults, totalThreads *sizeof(float));
    hipMalloc((void **)&devResults, totalThreads *sizeof(float));
    hipMemset(devResults, 0, totalThreads *sizeof(float));

	kanKernel<<<blocks,threads>>>(devResults);

	hipMemcpy(hostResults, devResults, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0;i<totalThreads;i++){
		std::cout << "Random no: " << hostResults[i] << std::endl;
	}
}
