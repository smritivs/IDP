#include "hip/hip_runtime.h"
#include "kan.h"

__global__ void randKernel(float *y){

	// initialise random state
	hiprandStatePhilox4_32_10_t localRandState;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(seed, idx, 0, &localRandState);

    y[idx] = hiprand_uniform(&localRandState);
	// input x is of size batch_size,input_dim
	// output y is of size batch_size, output_dim

	// first x * k


}

__global__ void kanKernel(
	float ***x,
	float ***fourierCoeffs,
	float **y
	){

	// check id value bounds

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int idz = threadIdx.z + blockIdx.z * blockDim.z;
	// input x is of size batch_size, input_dim
	// output y is of size batch_size, output_dim

	// x*k
	float x_element = x[idz][idy][0];

	__syncthreads();

	x[idz][idy][idx] = x_element * idz;


	float cosTerms[batchSize][inputDim][numHarmonics];
	float sinTerms[batchSize][inputDim][numHarmonics];

	float trigInp = x[idz][idy][idx];
	float cosRes = 0;
	float sinRes = 0;

	sincosf(trigInp,&sinRes,&cosRes);

	cosTerms[idz][idy][idx] = cosRes;
	sinTerms[idz][idy][idx] = sinRes;

	// second bounds check for output dims

	float yCos[batchSize][outputDim];
	float ySin[batchSize][outputDim];

	// optimize
	for(int i=0;i<inputDim;i++){
		for(int j=0;j<numHarmonics;j++){
			yCos[idz][idy] += cosTerms[idz][i][j] * fourierCoeffs[idy][i][j];
			ySin[idz][idy] += sinTerms[idz][i][j] * fourierCoeffs[idy][i][j];
		}
	}

	y[idz][idy] = yCos[idz][idy] + ySin[idz][idy] + bias;
}

int main(){
	int blocks = 10000;
	int threads = 1024;
	int totalThreads = blocks*threads;

	float *hostResults = (float *)calloc(totalThreads, sizeof(float));
	float *devResults = (float *)calloc(totalThreads, sizeof(float));
    /* Allocate space for results on device */
    hipMalloc((void **)&devResults, totalThreads *sizeof(float));
    hipMalloc((void **)&devResults, totalThreads *sizeof(float));
    hipMemset(devResults, 0, totalThreads *sizeof(float));

	randKernel<<<blocks,threads>>>(devResults);

	hipMemcpy(hostResults, devResults, totalThreads * sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0;i<totalThreads;i++){
		std::cout << "Random no: " << hostResults[i] << std::endl;
	}
}
